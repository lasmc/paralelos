
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

using namespace std;
__global__
void matMultKernel(float *d_M, float *d_N, float *d_P, int Width){
  int Row = blockIdx.y*blockDim.y + threadIdx.y;
  int Col = blockIdx.x*blockDim.x + threadIdx.x;
  int k = 0;
  if(Row < Width && Col < Width){
      float Pvalue = 0;
      for(k = 0; k < Width; ++k){
          Pvalue += d_M[Row*Width + k] * d_N[k*Width+Col];
      }
      d_P[Row*Width+Col] = Pvalue;
  }
}

void matMult(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  dim3 dimGrid(ceil(n/1024.0),ceil(n/1024.0),1);
  dim3 dimBlock(1024,1024,1);  /*Dimensiónde 2 X 2, eje Z desactivado con 1*/
  matMultKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
  n = 10000;
  /******************Time Variables*********************************/
  clock_t time;
  cout<<"El Tam de la matriz Cuadrada es: "<<n<<" X "<<n<<endl;
  h_A = new float[n*n];
  h_B = new float[n*n];
  h_C = new float[n*n];

  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++)
      h_A[i*n+j] = 1;
  }
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++)
      h_B[i*n+j] = 1;
  }
  time = clock();
  matMult(h_A,h_B,h_C,n);
  time = clock() - time;
  cout<<"El Tiempo  es: "<<(((float)time)/CLOCKS_PER_SEC)<<endl;


  /*for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
        cout<<h_C[i*n+j]<<" ; ";
    }
    cout<<endl;
  }
    cout<<endl;*/
  return 0;
}