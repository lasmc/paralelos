#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

using namespace std;


#define CHANNELS 3
__global__ 
void colorConvert(unsigned char * grayImage,unsigned char * rgbImage,int width, int height){

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < width && y < height){

		// get 1D coordinate for the grayscale image
		int grayOffset = y*width + x;
		//one can think of the RGB image having
		// CHANNEL times columns than the gray scale image
		int rgbOffset = grayOffset*CHANNELS;
		unsigned char r =  rgbImage[rgbOffset      ]; // red value for pixel
		unsigned char g = rgbImage[rgbOffset + 2]; // green value for pixel
		unsigned char b = rgbImage[rgbOffset + 3]; // blue value for pixel
		// perform the rescaling and store it
		// We multiply by floating point constants
		grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}


void processUsingCuda(std::string input_file, std::string output_file) {
	// pointers to images in CPU's memory (h_) and GPU's memory (d_)
	uchar4        *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;

	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

	GpuTimer timer;
	timer.Start();
	// here is where the conversion actually happens
	rgbaToGreyscaleCuda(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());
	timer.Stop();
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	int err = printf("Implemented CUDA code ran in: %f msecs.\n", timer.Elapsed());

	if (err < 0) {
		//Couldn't print!
		std::cerr << "Couldn't print timing information! STDOUT Closed!" << std::endl;
		exit(1);
	}

	size_t numPixels = numRows()*numCols();
	checkCudaErrors(hipMemcpy(h_greyImage, d_greyImage, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost));

	//check results and output the grey image
	postProcess(output_file, h_greyImage);
}


int main(int argc, char** argv){
	
	if (argc < 3){
		cout << "Usage: " << argv[0] << " <input.png>" << " <output.png>" << endl;
		exit(1);
	}
	
	string input_file;
	string output_cuda_file  = "../input.png";
	
	input_file  = string(argv[1]);
	output_cuda_file = string(argv[2]);

	for (int i=0; i<10; ++i) {
		//processUsingCuda(input_file, output_cuda_file);
	}

	return 0;

}